#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include "gputimer.h"

#define grid_size 4
#define total_entries (grid_size * grid_size)

#define top 0
#define right 1
#define bottom 2
#define left 3

#define rho 0.5
#define eta 2E-4
#define G 0.75

struct element_s {
    float u;
    float u1;
    float u2;
};

// Create a grid
int createMatrix(struct element_s *matrix, int hit) {
    //struct element_s matrix[total_entries] = {0};
    for (int i = 0; i < total_entries; i++) {
        matrix[i].u = 0;
        matrix[i].u1 = 0;
        matrix[i].u2 = 0;
    }
    if (hit == 1) {
        int hit_idx = grid_size * (grid_size / 2) + (grid_size / 2);
        matrix[hit_idx].u = 1;
        matrix[hit_idx].u1 = 1;
    }
    return 0;
}
__device__ int get_array_idx(int i, int j) {
    return (grid_size * i + j);
}

__device__ int idx_to_ij(int idx, int position) {
    int i = idx / grid_size;
    int j = idx % grid_size;

    switch(position){
        case 0:
            return (grid_size * (i - 1) + j);

        case 1:
            return (grid_size * i + (j + 1));

        case 2:
            return (grid_size * (i + 1) + j);

        case 3:
            return (grid_size * i + (j - 1));

        default:
            return 0;
    }
}

__device__ __host__ int store_prev(struct element_s *entry, float new_value) {
    entry->u2 = entry->u1;
    entry->u1 = entry->u;
    entry->u = new_value;

    return 0;
}

int printMatrix(struct element_s *matrix, int u_idx) {
    for (int i = 0; i < grid_size; i++) {
        for (int j = 0; j < grid_size; j++) {
        	if (u_idx == 0) {
        		printf("(%d, %d) = %.6f\t", i, j, matrix[grid_size * i + j].u);
        	}
            else if (u_idx == 1) {
            	printf("(%d, %d) = %.6f\t", i, j, matrix[grid_size * i + j].u1);
            }
            else if (u_idx == 2) {
            	printf("(%d, %d) = %.6f\t", i, j, matrix[grid_size * i + j].u2);
            }
        }
        printf("\n");
    }
    printf("\n");
    return 0;
}

int printOutput(struct element_s *matrix) {
    int output_idx = grid_size * (grid_size / 2) + (grid_size / 2);
    //printf("(%d, %d) %.6f\n", grid_size / 2, grid_size / 2, matrix[output_idx].u);
    printf("%.6f\n", matrix[output_idx].u);

    return 0;
}

__device__ int printLocal(struct element_s element) {
    printf("u = %.6f\t", element.u);
    printf("u1 = %.6f\t", element.u1);
    printf("u2 = %.6f\n", element.u2);

    return 0;
}
__global__ void interior(struct element_s * d_out, struct element_s * d_in ){
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    //printf("interior: thread (%d,%d) in block (%d,%d): idx = %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, idx);
        
    if (!(idx % grid_size == 0 || idx % grid_size == (grid_size - 1) ||idx < get_array_idx(0, grid_size - 1) || idx > get_array_idx(grid_size - 2, grid_size - 1))){
        //interior elements
        float temp;
        temp = (rho*(d_in[idx_to_ij(idx, top)].u1 + d_in[idx_to_ij(idx, bottom)].u1 + d_in[idx_to_ij(idx, left)].u1 + d_in[idx_to_ij(idx, right)].u1 - 4 * d_in[idx].u1)
                    + 2 * d_in[idx].u1 - (1 - eta) * d_in[idx].u2) / (1 + eta);
        store_prev(&d_in[idx], temp);
        printf("interior idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
        
    }
    d_out[idx] = d_in[idx];
    
}
__global__ void edge(struct element_s * d_out, struct element_s * d_in){
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    //printf("edge: thread (%d,%d) in block (%d,%d): idx = %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, idx);
        
    if (idx == get_array_idx(0, 0) || idx == get_array_idx(0, grid_size - 1) ||
                idx == get_array_idx(grid_size - 1, 0) || idx == get_array_idx(grid_size - 1, grid_size - 1)) {
        //ignore elements located at the corner
    }
    //top edge
    else if (idx < get_array_idx(0, grid_size)) {
        float temp;
        temp = G * d_in[idx + grid_size].u;
        store_prev(&d_in[idx], temp);
        printf("edge idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
    }
    //bottom edge
    else if (idx > get_array_idx(grid_size - 2, grid_size - 1)) {
        float temp;
        temp = G * d_in[idx - grid_size].u;
        store_prev(&d_in[idx], temp);
        printf("edge idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
    }
    //left edge
    else if (idx % grid_size == 0) {
        float temp;
        temp = G * d_in[idx + 1].u;
        store_prev(&d_in[idx], temp);
        printf("edge idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
    }
    //right edge
    else if (idx % grid_size == (grid_size - 1)) {
        float temp;
        temp = G * d_in[idx - 1].u;
        store_prev(&d_in[idx], temp);
        printf("edge idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
    }

    d_out[idx] = d_in[idx];
}
__global__ void corner(struct element_s * d_out, struct element_s * d_in){
    int idx = blockIdx.x * blockDim.x * blockDim.y + threadIdx.y * blockDim.x + threadIdx.x;
    //printf("corner: thread (%d,%d) in block (%d,%d): idx = %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, idx);

    //top left corner
    if (idx == get_array_idx(0, 0)) {
        float temp;
        temp = G * d_in[idx + grid_size].u;
        store_prev(&d_in[idx], temp);
        //printf("corner: thread (%d,%d) in block (%d,%d): idx = %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, idx);
        printf("corner idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
    }
    //top right and bottom right
    else if (idx == get_array_idx(0, grid_size - 1) || idx == get_array_idx(grid_size - 1, grid_size - 1)) {
        float temp;
        temp = G * d_in[idx - 1].u;
        store_prev(&d_in[idx], temp);
        //printf("corner: thread (%d,%d) in block (%d,%d): idx = %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, idx);
        printf("corner idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
    }
    //bottom left
    else if (idx == get_array_idx(grid_size - 1, 0)) {
        float temp;
        temp = G * d_in[idx - grid_size].u;
        store_prev(&d_in[idx], temp);
        //printf("corner: thread (%d,%d) in block (%d,%d): idx = %d\n", threadIdx.x, threadIdx.y, blockIdx.x, blockIdx.y, idx);
        printf("corner idx:%d, d_in[idx].u:%.6f\n",idx,d_in[idx].u);
    }
    
    d_out[idx] = d_in[idx];
}
int main(int argc, char **argv) {
    const int ARRAY_BYTES = total_entries * sizeof(element_s);

	if(argv[1] == NULL)
    {
        printf("Missing argument for iteration\n");
        return 0;
    }
    int T = atoi(argv[1]);
    // initialize the input array on the host
    struct element_s h_array[total_entries] = {0};
    createMatrix(h_array, 1);   

    // declare GPU memory pointers
    struct element_s * d_in;
    struct element_s * d_out;

    // allocate GPU memory
    hipMalloc(&d_in, ARRAY_BYTES);
    hipMalloc(&d_out, ARRAY_BYTES);

    GpuTimer timer;

    dim3 dimBlock(grid_size, grid_size, 1);
    dim3 dimGrid(1, 1, 1);
    while (T --> 0) {
        //interior
        // transfer the array to the GPU
        hipMemcpy(d_in, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);
        // launch the kernel to update interior elements
        timer.Start();
        interior<<<dimGrid, dimBlock>>>(d_out, d_in);
        timer.Stop();
        // copy back the result array to the CPU
        hipMemcpy(h_array, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost); 

        //edge
        // transfer the updated array to the GPU
        hipMemcpy(d_in, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);
        // launch the kernel
        timer.Start();
        edge<<<dimGrid, dimBlock>>>(d_out, d_in);
        timer.Stop();
        // copy back the result array to the CPU
        hipMemcpy(h_array, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost); 

        //corner
        // transfer the updated array to the GPU
        hipMemcpy(d_in, h_array, ARRAY_BYTES, hipMemcpyHostToDevice);
        // launch the kernel
        timer.Start();
        corner<<<dimGrid, dimBlock>>>(d_out, d_in);
        timer.Stop();
        // copy back the result array to the CPU
        hipMemcpy(h_array, d_out, ARRAY_BYTES, hipMemcpyDeviceToHost); 
        
        for (int i = 0; i < total_entries; i++){
            store_prev(&h_array[i], h_array[i].u);
        }  

        printMatrix(h_array, 0);
        printOutput(h_array);

    }
    printf("Time elapsed = %g ms\n", timer.Elapsed());
    hipFree(d_in);
    hipFree(d_out);

    return 0;
}

